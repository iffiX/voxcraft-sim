#include "hip/hip_runtime.h"
//
// Created by iffi on 11/6/22.
//

#include "vx3_voxelyze_kernel_manager.cuh"
#include <boost/format.hpp>

using namespace std;
using namespace boost;

VX3_VoxelyzeKernel
VX3_VoxelyzeKernelManager::createKernelFromConfig(const VX3_Config &config,
                                                  const hipStream_t &stream) {

    for (auto &material_config : config.palette.materials) {
        if (material_config.material_id == 0) {
            // The default "empty" material
            ictx.voxel_materials.emplace_back();
        } else {
            addVoxelMaterial(material_config, (int)ictx.voxel_materials.size(),
                             config.lattice.lattice_dim, config.bond_damping_z,
                             config.col_damping_z, config.slow_damping_z);
        }
    }
    // voxels and links (links are added when creating voxels)
    addVoxels(config.structure, config.lattice.lattice_dim);

    // Initialize
    for (auto &voxel_material : ictx.voxel_materials)
        voxel_material.init(ictx);
    for (auto &link_material : ictx.link_materials)
        link_material.init(ictx);
    for (auto &voxel : ictx.voxels)
        voxel.init(ictx);
    for (auto &link : ictx.links)
        link.init(ictx);

    // Copy data to the voxelyze kernel
    VX3_VoxelyzeKernel kernel;
    kernel.ctx.link_materials.fill(ictx.link_materials, stream);
    kernel.ctx.voxel_materials.fill(ictx.voxel_materials, stream);
    kernel.ctx.links.fill(ictx.links, stream);
    kernel.ctx.voxels.fill(ictx.voxels, stream);

    kernel.vox_size = config.lattice.lattice_dim;
    kernel.dt_frac = config.dt_frac;

    // In VXA.Simulator.StopCondition
    setMathExpression(kernel.stop_condition, config.stop_condition);
    setMathExpression(kernel.result_start_condition, config.result_start_condition);
    setMathExpression(kernel.result_end_condition, config.result_end_condition);

    // In VXA.Simulator.RecordHistory
    kernel.record_step_size = config.record_step_size;
    kernel.record_link = config.record_link;
    kernel.record_voxel = config.record_voxel;

    // In VXA.Simulator.AttachDetach
    kernel.enable_attach = config.enable_attach;
    kernel.enable_detach = config.enable_detach;
    kernel.enable_collision = config.enable_collision;
    kernel.bounding_radius = config.bounding_radius;
    kernel.watch_distance = config.watch_distance;
    kernel.safety_guard = config.safety_guard;
    for (size_t i = 0; i < 5; i++)
        setMathExpression(kernel.attach_conditions[i], config.attach_conditions[i]);

    // In VXA.Simulator.ForceField
    setMathExpression(kernel.force_field.x_force_field, config.x_force_field);
    setMathExpression(kernel.force_field.y_force_field, config.y_force_field);
    setMathExpression(kernel.force_field.z_force_field, config.z_force_field);

    // In VXA.Simulator
    setMathExpression(kernel.fitness_function, config.fitness_function);
    kernel.max_dist_in_voxel_lengths_to_count_as_pair =
        config.max_dist_in_voxel_lengths_to_count_as_pair;
    kernel.save_position_of_all_voxels = config.save_position_of_all_voxels;
    kernel.enable_cilia = config.enable_cilia;
    kernel.enable_signals = config.enable_signals;

    // In VXA.Environment.Gravity
    kernel.grav_enabled = config.grav_enabled;
    kernel.floor_enabled = config.floor_enabled;
    kernel.grav_acc = config.grav_acc;

    // In VXA.Environment.Thermal
    kernel.enable_vary_temp = config.enable_vary_temp;
    kernel.temp_amplitude = config.temp_amplitude;
    kernel.temp_period = config.temp_period;

    // Collect target voxel indices
    auto tmp_target_indices = new Vindex[ictx.voxels.size()];
    Vsize target_num = 0;
    for (Vindex vox = 0; vox < ictx.voxels.size(); vox++) {
        if (ictx.voxel_materials[ictx.voxels[vox].voxel_material].is_target) {
            tmp_target_indices[target_num++] = vox;
        }
    }
    VcudaMallocAsync(&kernel.d_target_indices, sizeof(Vindex) * target_num, stream);
    VcudaMemcpyAsync(kernel.d_target_indices, tmp_target_indices,
                     sizeof(Vindex) * target_num, hipMemcpyHostToDevice, stream);
    // Make sure all host data are transferred
    VcudaStreamSynchronize(stream);
    delete[] tmp_target_indices;
    kernel.target_num = target_num;

    return kernel;
}

void VX3_VoxelyzeKernelManager::freeKernel(VX3_VoxelyzeKernel &kernel,
                                           const hipStream_t &stream) {
    kernel.ctx.link_materials.free(stream);
    kernel.ctx.voxel_materials.free(stream);
    kernel.ctx.links.free(stream);
    kernel.ctx.voxels.free(stream);
    VcudaFreeAsync(kernel.d_target_indices, stream);
    VcudaFreeAsync(kernel.d_steps, stream);
    VcudaFreeAsync(kernel.d_time_points, stream);
    VcudaFreeAsync(kernel.d_link_record, stream);
    VcudaFreeAsync(kernel.d_voxel_record, stream);
    // Make sure all free actions are finished
    VcudaStreamSynchronize(stream);
}

void VX3_VoxelyzeKernelManager::addVoxelMaterial(
    const VX3_PaletteMaterialConfig &material_config, int material_id, Vfloat lattice_dim,
    Vfloat internal_damping, Vfloat collision_damping, Vfloat global_damping) {
    VX3_VoxelMaterial mat;

    // Base material attributes
    mat.r = int(round(material_config.r * 255));
    mat.g = int(round(material_config.g * 255));
    mat.b = int(round(material_config.b * 255));
    mat.a = int(round(material_config.a * 255));

    mat.material_id = material_id;
    mat.fixed = material_config.fixed;
    mat.sticky = material_config.sticky;
    mat.cilia = material_config.cilia;
    mat.linear = material_config.mat_model == VX3_PaletteMaterialConfig::MAT_LINEAR;

    mat.E = material_config.elastic_mod;
    mat.nu = material_config.poissons_ratio;
    mat.rho = material_config.density;
    mat.alpha_CTE = material_config.CTE;
    mat.fail_stress = material_config.fail_stress;
    mat.u_static = material_config.u_static;
    mat.u_kinetic = material_config.u_dynamic;
    mat.zeta_internal = internal_damping;
    mat.zeta_collision = collision_damping;
    mat.zeta_global = global_damping;

    mat.is_target = material_config.is_target;
    mat.is_measured = material_config.is_measured;
    mat.is_pace_maker = material_config.is_pace_maker;
    mat.pace_maker_period = material_config.pace_maker_period;
    mat.signal_value_decay = material_config.signal_value_decay;
    mat.signal_time_delay = material_config.signal_time_delay;
    mat.inactive_period = material_config.inactive_period;

    // Voxel material attributes
    mat.nom_size = lattice_dim;

    ictx.voxel_materials.emplace_back(mat);
}

Vindex VX3_VoxelyzeKernelManager::addOrGetLinkMaterial(Vindex voxel1_material_index,
                                                       Vindex voxel2_material_index) {
    auto link_material_index = voxel_materials_to_link_material_index.find(
        {voxel1_material_index, voxel2_material_index});
    if (link_material_index != voxel_materials_to_link_material_index.end())
        return link_material_index->second;
    VX3_LinkMaterial link_material;
    link_material.vox1_mat = voxel1_material_index;
    link_material.vox2_mat = voxel2_material_index;
    Vindex new_link_material_index = ictx.link_materials.size();
    voxel_materials_to_link_material_index[{
        voxel1_material_index, voxel2_material_index}] = new_link_material_index;

    ictx.link_materials.emplace_back(link_material);
    return new_link_material_index;
}

void VX3_VoxelyzeKernelManager::addVoxels(const VX3_StructureConfig &structure_config,
                                          Vfloat vox_size) {
    for (short z = 0; z < structure_config.z_voxels; z++) {
        for (short y = 0; y < structure_config.y_voxels; y++) {
            for (short x = 0; x < structure_config.x_voxels; x++) {
                int idx_1d = index3DToIndex1D(x, y, z, structure_config.x_voxels,
                                              structure_config.y_voxels);
                // Skip empty voxels
                if (structure_config.data[idx_1d] == 0)
                    continue;
                auto voxel = VX3_Voxel();
                voxel.index_x = x;
                voxel.index_y = y;
                voxel.index_z = z;
                voxel.position.x = (Vfloat)x * vox_size;
                voxel.position.y = (Vfloat)y * vox_size;
                voxel.position.z = (Vfloat)z * vox_size;
                voxel.initial_position = voxel.position;
                voxel.voxel_material = (Vindex)(structure_config.data[idx_1d]);
                voxel.amplitude = structure_config.amplitudes[idx_1d];
                voxel.frequency = structure_config.frequencies[idx_1d];
                voxel.phase_offset = structure_config.phase_offsets[idx_1d];
                voxel.base_cilia_force = structure_config.base_cilia_force[idx_1d];
                voxel.shift_cilia_force = structure_config.shift_cilia_force[idx_1d];
                for (unsigned int &link : voxel.links) {
                    link = NULL_INDEX;
                }
                coordinate_to_voxel_index[index3DToCoordinate(x, y, z)] =
                    ictx.voxels.size();
                ictx.voxels.emplace_back(voxel);
                // add any possible links utilizing this voxel
                for (int i = 0; i < 6; i++) {
                    addLink(x, y, z, (LinkDirection)i);
                }
            }
        }
    }
}

void VX3_VoxelyzeKernelManager::addLink(short x, short y, short z,
                                        LinkDirection direction) {
    auto voxel1_coords = index3DToCoordinate(x, y, z);
    auto voxel2_coords = index3DToCoordinate(x + (short)xIndexVoxelOffset(direction),
                                             y + (short)yIndexVoxelOffset(direction),
                                             z + (short)zIndexVoxelOffset(direction));
    if (coordinate_to_voxel_index.find(voxel1_coords) ==
            coordinate_to_voxel_index.end() ||
        coordinate_to_voxel_index.find(voxel2_coords) == coordinate_to_voxel_index.end())
        return;

    // Since a link can only be added when voxel2 pre-exists, and
    // voxel2 cannot create a link before voxel1 is added, there is
    // no need to check duplicate links because all links are unique
    VX3_Link link;
    link.axis = linkDirectionToAxis(direction);
    // Make sure that link is always pointed to +X, +Y and +Z direction
    // from negative voxel to positive voxel
    bool reverse_order = false;
    if (link.axis == X_AXIS)
        reverse_order = direction == X_NEG;
    else if (link.axis == Y_AXIS)
        reverse_order = direction == Y_NEG;
    else if (link.axis == Z_AXIS)
        reverse_order = direction == Z_NEG;

    if (reverse_order) {
        link.voxel_neg = coordinate_to_voxel_index[voxel2_coords];
        link.voxel_pos = coordinate_to_voxel_index[voxel1_coords];
    } else {
        link.voxel_neg = coordinate_to_voxel_index[voxel1_coords];
        link.voxel_pos = coordinate_to_voxel_index[voxel2_coords];
    }
    link.link_material = addOrGetLinkMaterial(ictx.voxels[link.voxel_neg].voxel_material,
                                              ictx.voxels[link.voxel_pos].voxel_material);

    Vindex new_link_index = ictx.links.size();
    ictx.links.emplace_back(link);

    // Store link reference to voxels
    ictx.voxels[link.voxel_neg].links[direction] = new_link_index;
    ictx.voxels[link.voxel_pos].links[oppositeLinkDirection(direction)] = new_link_index;
}

void VX3_VoxelyzeKernelManager::setMathExpression(
    VX3_MathTreeToken *tokens, const VX3_Config::VX3_MathTreeExpression &expr) {
    if (expr.size() > VX3_MATH_TREE_MAX_EXPRESSION_TOKENS)
        throw std::invalid_argument("Math expression size too large");
    for (size_t i = 0; i < expr.size(); i++)
        tokens[i] = expr[i];
}

inline uint64_t VX3_VoxelyzeKernelManager::index3DToCoordinate(short x, short y,
                                                               short z) const {
    return uint64_t(x) << 32 | uint64_t(y) << 16 | uint64_t(z);
}

inline int VX3_VoxelyzeKernelManager::index3DToIndex1D(int x, int y, int z, int x_size,
                                                       int y_size) const {
    return z * (x_size * y_size) + y * x_size + x;
}