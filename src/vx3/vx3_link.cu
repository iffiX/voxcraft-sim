#include "hip/hip_runtime.h"
#include "vx3/vx3_link_material.h"
#include "vx3/vx3_voxel.h"
#include "vx3/vx3_voxel_material.h"
#include "vx3/vx3_voxelyze_kernel.cuh"
#include "vx3_link.h"

// Must include this at end because we need to define
// reflection macros in vx3_link, vx3_voxel, etc. first
#include "vx3/vx3_context.h"

/**
 * Parameter naming:
 * _some_var: is temporary variable which stores new states that will be used
 *  to update the member variable;
 * some_var: is local copy of the existing states that will not be used
 *  to update the member variable;
 */

void VX3_Link::init(const VX3_InitContext &ictx) {
    bool_states = 0;
    force_neg = Vec3f();
    force_pos = Vec3f();
    moment_neg = Vec3f();
    moment_pos = Vec3f();
    strain = 0;
    max_strain = 0;
    strain_offset = 0;

    Vfloat E_pos = ictx.voxel_materials[ictx.voxels[voxel_pos].voxel_material].E;
    Vfloat E_neg = ictx.voxel_materials[ictx.voxels[voxel_neg].voxel_material].E;
    strain_ratio = E_pos / E_neg;

    pos2 = Vec3f();
    angle1v = Vec3f();
    angle2v = Vec3f();
    angle1 = Quat3f();
    angle2 = Quat3f();
    is_small_angle = false;
    current_rest_length = 0;
    current_transverse_area = 0;
    current_transverse_strain_sum = 0;
    axial_stress = 0;
    is_new_link = 0;
    is_detached = false;
    removed = false;
}

__device__ Vfloat VX3_Link::axialStrain(const VX3_Context &ctx, Vindex link,
                                        bool positive_end) {
    Vfloat s = L_G(strain);
    Vfloat sr = L_G(strain_ratio);
    return positive_end ? 2.0f * s * sr / (1.0f + sr) : 2.0f * s / (1.0f + sr);
}

__device__ bool VX3_Link::isYielded(const VX3_Context &ctx, Vindex link) {
    return VX3_Material::isYielded<false>(ctx, L_G(max_strain), L_G(link_material));
}

__device__ bool VX3_Link::isFailed(const VX3_Context &ctx, Vindex link) {
    return VX3_Material::isFailed<false>(ctx, L_G(link_material), L_G(max_strain));
}

__device__ void VX3_Link::updateRestLength(VX3_Context &ctx, Vindex link) {
    // update rest length according to temperature of both end
    auto axis = L_G(axis);
    auto neg_base_size = VX3_Voxel::baseSize(ctx, L_G(voxel_neg), L_G(axis));
    auto pos_base_size = VX3_Voxel::baseSize(ctx, L_G(voxel_neg), L_G(axis));
    L_S(current_rest_length, 0.5f * (neg_base_size + pos_base_size));
}

__device__ void VX3_Link::updateTransverseInfo(VX3_Context &ctx, Vindex link) {
    L_S(current_transverse_area,
        0.5f * (VX3_Voxel::transverseArea(ctx, L_G(voxel_neg), L_G(axis)) +
                VX3_Voxel::transverseArea(ctx, L_G(voxel_pos), L_G(axis))));
    L_S(current_transverse_strain_sum,
        0.5f * (VX3_Voxel::transverseStrainSum(ctx, L_G(voxel_neg), L_G(axis)) +
                VX3_Voxel::transverseStrainSum(ctx, L_G(voxel_pos), L_G(axis))));
}

// updates pos2, angle1, angle2, and smallAngle
__device__ Quat3f VX3_Link::orientLink(VX3_Context &ctx, Vindex link) {
    Vec3f _pos2 = V_G(L_G(voxel_pos), position) - V_G(L_G(voxel_neg), position);
    // digit truncation happens here...
    _pos2 = toAxisX(L_G(axis), _pos2);

    Quat3f _angle1 = V_G(L_G(voxel_neg), orientation);
    _angle1 = toAxisX(L_G(axis), _angle1);

    Quat3f _angle2 = V_G(L_G(voxel_pos), orientation);
    _angle2 = toAxisX(L_G(axis), _angle2);

    // keep track of the total rotation of this bond (after toAxisX())
    Quat3f total_rot = _angle1.conjugate();
    _pos2 = total_rot.rotateVec3D(_pos2);
    _angle1 = Quat3f(); // zero for now...
    _angle2 = total_rot * _angle2;

    // small angle approximation?
    Vfloat curr_rest_length = L_G(current_rest_length);
    Vfloat small_turn = (abs(_pos2.z) + abs(_pos2.y)) / _pos2.x;
    Vfloat extend_perc = abs(1 - _pos2.x / curr_rest_length);
    bool _is_small_angle = L_G(is_small_angle);
    if (!_is_small_angle && small_turn < SA_BOND_BEND_RAD &&
        extend_perc < SA_BOND_EXT_PERC) {
        _is_small_angle = true;
        setBoolState(ctx, link, LOCAL_VELOCITY_VALID, false);
    } else if (_is_small_angle && small_turn > HYSTERESIS_FACTOR * SA_BOND_BEND_RAD ||
               extend_perc > HYSTERESIS_FACTOR * SA_BOND_EXT_PERC) {
        _is_small_angle = false;
        setBoolState(ctx, link, LOCAL_VELOCITY_VALID, false);
    }

    if (_is_small_angle) {
        // Align so Angle1 is all zeros

        // only valid for small angles
        _pos2.x -= curr_rest_length;
    } else {
        // Large angle. Align so that Pos2.y, Pos2.z are zero.

        // get the angle to align Pos2 with the X axis
        _angle1.fromAngleToPosX(_pos2);
        total_rot = _angle1 * total_rot; // update our total rotation to reflect this
        _angle2 = _angle1 * _angle2;     // rotate angle2
        _pos2 = Vec3f(_pos2.length() - curr_rest_length, 0, 0);
    }
    // State updates
    L_S(is_small_angle, _is_small_angle);
    L_S(angle1, _angle1);
    L_S(angle2, _angle2);
    L_S(pos2, _pos2);
    L_S(angle1v, _angle1.toRotationVector());
    L_S(angle2v, _angle1.toRotationVector());
    Vec3f a1v = L_G(angle1v);
    Vec3f a2v = L_G(angle2v);

    // assert non QNAN
    // assert(not isnan(a1v.x) && not isnan(a1v.y) && not isnan(a1v.z));
    // assert(not isnan(a2v.x) && not isnan(a2v.y) && not isnan(a2v.z));

    return total_rot;
}

__device__ void VX3_Link::updateForces(VX3_Context &ctx, Vindex link) {
    // remember the positions/angles from last timestamp to
    // calculate velocity
    Vec3f old_pos2 = L_G(pos2);
    Vec3f old_angle1v = L_G(angle1v);
    Vec3f old_angle2v = L_G(angle2v);

    // sets pos2, angle1, angle2
    orientLink(ctx, link);

    // deltas for local damping. velocity at center
    // is half the total velocity
    Vec3f new_pos2 = L_G(pos2);
    Vec3f new_angle1v = L_G(angle1v);
    Vec3f new_angle2v = L_G(angle2v);
    Vec3f d_pos2 = 0.5 * (new_pos2 - old_pos2);
    Vec3f d_angle1 = 0.5 * (new_angle1v - old_angle1v);
    Vec3f d_angle2 = 0.5 * (new_angle2v - old_angle2v);

    // if volume effects..
    if (!VX3_Material::isXyzIndependent<false>(ctx, L_G(link_material)) ||
        L_G(current_transverse_strain_sum) != 0) {
        // current_transverse_strain_sum != 0 catches when we disable
        // poissons mid-simulation
        // updateTransverseInfo();
    }
    Vfloat _axial_stress = updateStrain(ctx, link, new_pos2.x / L_G(current_rest_length));
    L_S(axial_stress, _axial_stress);
    if (isFailed(ctx, link)) {
        L_S(force_neg, Vec3f());
        L_S(force_pos, Vec3f());
        L_S(force_neg, Vec3f());
        L_S(force_pos, Vec3f());
        return;
    }

    // local copies
    Vindex link_mat = L_G(link_material);
    Vfloat b1 = LM_G(link_mat, b1), b2 = LM_G(link_mat, b2), b3 = LM_G(link_mat, b3),
           a2 = LM_G(link_mat, a2);

    // Beam equations. All relevant terms are here, even though some are zero
    // for small angle and others are zero for large angle (profiled as
    // negligible performance penalty)

    // Use Curstress instead of -a1*Pos2.x to account for non-linear deformation
    Vec3f _force_neg = Vec3f(_axial_stress * L_G(current_transverse_area),
                             b1 * new_pos2.y - b2 * (new_angle1v.z + new_angle2v.z),
                             b1 * new_pos2.z + b2 * (new_angle1v.y + new_angle2v.y));

    Vec3f _moment_neg = Vec3f(a2 * (new_angle2v.x - new_angle1v.x),
                              -b2 * new_pos2.z - b3 * (2 * new_angle1v.y + new_angle2v.y),
                              b2 * new_pos2.y - b3 * (2 * new_angle1v.z + new_angle2v.z));
    Vec3f _moment_pos = Vec3f(a2 * (new_angle1v.x - new_angle2v.x),
                              -b2 * new_pos2.z - b3 * (new_angle1v.y + 2 * new_angle2v.y),
                              b2 * new_pos2.y - b3 * (new_angle1v.z + 2 * new_angle2v.z));
    L_S(force_neg, _force_neg);
    L_S(force_pos, -_force_neg);
    L_S(moment_neg, _moment_neg);
    L_S(moment_pos, _moment_pos);

    // local damping:
    if (getBoolState(ctx, link, LOCAL_VELOCITY_VALID)) {
        // if we don't have the basis for a good damping calculation,
        // don't do any damping.

        Vfloat sqA1 = LM_G(link_mat, sqA1), sqA2xIp = LM_G(link_mat, sqA2xIp),
               sqB1 = LM_G(link_mat, sqB1), sqB2xFMp = LM_G(link_mat, sqB2xFMp),
               sqB3xIp = LM_G(link_mat, sqB3xIp);

        Vec3f pos_calc(sqA1 * d_pos2.x,
                       sqB1 * d_pos2.y - sqB2xFMp * (d_angle1.z + d_angle2.z),
                       sqB1 * d_pos2.z + sqB2xFMp * (d_angle1.y + d_angle2.y));

        Vfloat voxel_neg_damping = VX3_Voxel::dampingMultiplier(ctx, L_G(voxel_neg));
        Vfloat voxel_pos_damping = VX3_Voxel::dampingMultiplier(ctx, L_G(voxel_pos));

        L_S(force_neg, L_G(force_neg) + voxel_neg_damping * pos_calc);
        L_S(force_pos, L_G(force_pos) + voxel_pos_damping * pos_calc);
        L_S(moment_neg,
            L_G(moment_neg) -
                0.5 * voxel_neg_damping *
                    Vec3f(-sqA2xIp * (d_angle2.x - d_angle1.x),
                          sqB2xFMp * d_pos2.z + sqB3xIp * (2 * d_angle1.y + d_angle2.y),
                          -sqB2xFMp * d_pos2.y +
                              sqB3xIp * (2 * d_angle1.z + d_angle2.z)));
        L_S(moment_pos,
            L_G(moment_pos) -
                0.5 * voxel_pos_damping *
                    Vec3f(sqA2xIp * (d_angle2.x - d_angle1.x),
                          sqB2xFMp * d_pos2.z + sqB3xIp * (d_angle1.y + 2 * d_angle2.y),
                          -sqB2xFMp * d_pos2.y +
                              sqB3xIp * (d_angle1.z + 2 * d_angle2.z)));

    } else {
        // we're good for next go-around unless something changes
        setBoolState(ctx, link, LOCAL_VELOCITY_VALID, true);
    }
    //	transform forces and moments to local voxel coordinates
    LinkAxis ax = L_G(axis);
    if (!L_G(is_small_angle)) {
        Quat3f ang1 = L_G(angle1);
        L_S(force_neg, toAxisOriginal(ax, ang1.rotateVec3DInv(L_G(force_neg))));
        L_S(moment_neg, toAxisOriginal(ax, ang1.rotateVec3DInv(L_G(moment_neg))));
    }

    Quat3f ang2 = L_G(angle2);
    L_S(force_neg, toAxisOriginal(ax, ang2.rotateVec3DInv(L_G(force_neg))));
    L_S(force_pos, toAxisOriginal(ax, ang2.rotateVec3DInv(L_G(force_pos))));
    L_S(moment_neg, toAxisOriginal(ax, ang2.rotateVec3DInv(L_G(moment_neg))));
    L_S(moment_pos, toAxisOriginal(ax, ang2.rotateVec3DInv(L_G(moment_pos))));

    if (L_G(is_new_link)) {
        // for debug
        L_S(force_neg, L_G(force_neg) * 0.01);
        L_S(force_pos, L_G(force_pos) * 0.01);
        L_S(moment_neg, L_G(moment_neg) * 0.01);
        L_S(moment_pos, L_G(moment_pos) * 0.01);
        L_S(is_new_link, L_G(is_new_link) - 1);
    }
}

__device__ float VX3_Link::updateStrain(VX3_Context &ctx, Vindex link,
                                        float axial_strain) {
    // int di = 0;
    // redundant?
    L_S(strain, axial_strain);

    Vindex link_material = L_G(link_material);
    if (LM_G(link_material, linear)) {
        // remember this maximum for easy reference
        if (axial_strain > L_G(max_strain))
            L_S(max_strain, axial_strain);
        return VX3_Material::stress<false>(ctx, link_material, axial_strain,
                                           L_G(current_transverse_strain_sum));
    } else {
        float return_stress;
        if (axial_strain > L_G(max_strain)) {
            // if new territory on the stress/strain curve

            // remember this maximum for easy reference
            L_S(max_strain, axial_strain);
            Vfloat max_strain = axial_strain;
            return_stress = VX3_Material::stress<false>(
                ctx, link_material, axial_strain, L_G(current_transverse_strain_sum));

            if (LM_G(link_material, nu) != 0.0f) {
                // precalculate strain offset for when we back off
                L_S(strain_offset,
                    max_strain -
                        VX3_Material::stress<false>(ctx, link_material, axial_strain) /
                            (LM_G(link_material, E_hat) * (1 - LM_G(link_material, nu))));
            } else {
                // Precalculate strain offset for when we back off
                L_S(strain_offset, max_strain - return_stress / LM_G(link_material, E));
            }

        } else {
            // backed off a non-linear material, therefore in linear
            // region.

            // treat the material as linear with
            // a strain offset according to the
            // maximum plastic deformation
            Vfloat relative_strain = axial_strain - L_G(strain_offset);

            if (LM_G(link_material, nu) != 0.0f)
                return_stress =
                    VX3_Material::stress<false>(ctx, link_material, relative_strain,
                                                L_G(current_transverse_strain_sum), true);
            else
                return_stress = LM_G(link_material, E) * relative_strain;
        }

        return return_stress;
    }
}

__device__ void VX3_Link::setBoolState(VX3_Context &ctx, Vindex link, LinkFlags flag,
                                       bool active) {
    active ? L_S(bool_states, L_G(bool_states) | (int)flag)
           : L_S(bool_states, L_G(bool_states) & ~(int)flag);
}

__device__ bool VX3_Link::getBoolState(const VX3_Context &ctx, Vindex link, LinkFlags flag) {
    return L_G(bool_states) & flag ? true : false;
}

__device__ Vfloat VX3_Link::strainEnergy(const VX3_Context &ctx, Vindex link) {
    Vec3f force_neg = L_G(force_neg);
    Vec3f moment_neg = L_G(moment_neg);
    Vec3f moment_pos = L_G(moment_pos);
    Vindex link_material = L_G(link_material);
    Vfloat a1 = LM_G(link_material, a1);
    Vfloat a2 = LM_G(link_material, a2);
    Vfloat b3 = LM_G(link_material, b3);
    return force_neg.x * force_neg.x / (2.0f * a1) +   // Tensile strain
           moment_neg.x * moment_neg.x / (2.0f * a2) + // Torsion strain
           (moment_neg.z * moment_neg.z - moment_neg.z * moment_pos.z +
            moment_pos.z * moment_pos.z) /
               (3.0f * b3) + // Bending Z
           (moment_neg.y * moment_neg.y - moment_neg.y * moment_pos.y +
            moment_pos.y * moment_pos.y) /
               (3.0f * b3); // Bending Y
}

__device__ Vfloat VX3_Link::axialStiffness(VX3_Context &ctx, Vindex link) {
    Vindex link_material = L_G(link_material);
    if (VX3_Material::isXyzIndependent<false>(ctx, link_material))
        return LM_G(link_material, a1);
    else {
        updateRestLength(ctx, link);
        updateTransverseInfo(ctx, link);
        return LM_G(link_material, E_hat) * L_G(current_transverse_area) /
               ((L_G(strain) + 1.0f) * L_G(current_rest_length)); // _a1;
    }
}